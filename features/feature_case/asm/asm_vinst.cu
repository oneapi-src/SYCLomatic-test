// ====------ asm_vinst.cu --------------------------------- *- CUDA -* ---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===--------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(ID, S, CMP)                                                      \
  {                                                                            \
    S;                                                                         \
    if (!(CMP)) {                                                              \
      return ID;                                                               \
    }                                                                          \
  }

// clang-format off
__device__ int vadd() {
  int i, b = 4, c = 5, d = 6;
  unsigned u;
  CHECK(1, asm("vadd.s32.u32.s32 %0, %1, %2;" : "=r"(i) : "r"(3), "r"(4)), i == 7);
  CHECK(2, asm("vadd.u32.u32.s32 %0, %1, %2;" : "=r"(u) : "r"(b), "r"(c)), u == 9);
  CHECK(3, asm("vadd.s32.u32.s32.sat %0, %1, %2;" : "=r"(i) : "r"(b), "r"(INT_MAX)), i == INT_MAX);
  // TODO: Need to keep the same behavior with asm.
  // CHECK(4, asm("vadd.u32.u32.s32.sat %0, %1, %2;" : "=r"(u) : "r"(UINT_MAX), "r"(INT_MAX)), u == 0x7FFFFFFEull);
  CHECK(5, asm("vadd.s32.u32.s32.sat.add %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-20), "r"(d)), i == -10);
  CHECK(6, asm("vadd.s32.u32.s32.sat.min %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(c), "r"(-20)), -20);
  CHECK(7, asm("vadd.s32.u32.s32.sat.max %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-33), "r"(9)), i == 9);

  return 0;
}

__device__ int vsub() {
  int i, b = 4, c = 5, d = 6;
  unsigned u;
  CHECK(1, asm("vsub.s32.u32.s32 %0, %1, %2;" : "=r"(i) : "r"(3), "r"(4)), i == -1);
  CHECK(2, asm("vsub.u32.u32.s32 %0, %1, %2;" : "=r"(u) : "r"(c), "r"(b)), u == 1);
  CHECK(3, asm("vsub.s32.u32.s32.sat %0, %1, %2;" : "=r"(i) : "r"(10), "r"(INT_MIN)), i == INT_MAX);
  CHECK(4, asm("vsub.u32.u32.s32.sat %0, %1, %2;" : "=r"(u) : "r"(0), "r"(1)), u == 0);
  CHECK(5, asm("vsub.s32.u32.s32.sat.add %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-20), "r"(d)), i == 30);
  CHECK(6, asm("vsub.s32.u32.s32.sat.min %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(c), "r"(-20)), -20);
  CHECK(7, asm("vsub.s32.u32.s32.sat.max %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-33), "r"(9)), i == 37);

  return 0;
}

__device__ int vabsdiff() {
  int i, b = 4, c = 5, d = 6;
  unsigned u;
  CHECK(1, asm("vabsdiff.s32.u32.s32 %0, %1, %2;" : "=r"(i) : "r"(3), "r"(4)), i == 1);
  CHECK(2, asm("vabsdiff.u32.u32.s32 %0, %1, %2;" : "=r"(u) : "r"(c), "r"(b)), u == 1);
  CHECK(3, asm("vabsdiff.s32.u32.s32.sat %0, %1, %2;" : "=r"(i) : "r"(10), "r"(INT_MIN)), i == INT_MAX);
  CHECK(4, asm("vabsdiff.u32.u32.s32.sat %0, %1, %2;" : "=r"(u) : "r"(0), "r"(1)), u == 1);
  CHECK(5, asm("vabsdiff.s32.u32.s32.sat.add %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-20), "r"(d)), i == 30);
  CHECK(6, asm("vabsdiff.s32.u32.s32.sat.min %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(c), "r"(-20)), -20);
  CHECK(7, asm("vabsdiff.s32.u32.s32.sat.max %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-33), "r"(9)), i == 37);

  return 0;
}

__device__ int vmin() {
  int i, b = 4, c = 5, d = 6;
  unsigned u;
  CHECK(1, asm("vmin.s32.u32.s32 %0, %1, %2;" : "=r"(i) : "r"(3), "r"(4)), i == 3);
  CHECK(2, asm("vmin.u32.u32.s32 %0, %1, %2;" : "=r"(u) : "r"(c), "r"(b)), u == 4);
  CHECK(3, asm("vmin.s32.u32.s32.sat %0, %1, %2;" : "=r"(i) : "r"(UINT_MAX), "r"(1)), i == 1);
  CHECK(4, asm("vmin.u32.u32.s32.sat %0, %1, %2;" : "=r"(u) : "r"(10), "r"(-1)), u == 0);
  CHECK(5, asm("vmin.s32.u32.s32.sat.add %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-20), "r"(d)), i == -14);
  CHECK(6, asm("vmin.s32.u32.s32.sat.min %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(c), "r"(-20)), -20);
  CHECK(7, asm("vmin.s32.u32.s32.sat.max %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-33), "r"(9)), i == 9);

  return 0;
}

__device__ int vmax() {
  int i, b = 4, c = 5, d = 6;
  unsigned u;
  CHECK(1, asm("vmax.s32.u32.s32 %0, %1, %2;" : "=r"(i) : "r"(3), "r"(4)), i == 4);
  CHECK(2, asm("vmax.u32.u32.s32 %0, %1, %2;" : "=r"(u) : "r"(c), "r"(b)), u == 5);
  CHECK(3, asm("vmax.s32.u32.s32.sat %0, %1, %2;" : "=r"(i) : "r"(UINT_MAX), "r"(1)), i == INT_MAX);
  CHECK(4, asm("vmax.u32.u32.s32.sat %0, %1, %2;" : "=r"(u) : "r"(UINT_MAX), "r"(1)), u == UINT_MAX);
  CHECK(5, asm("vmax.s32.u32.s32.sat.add %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-20), "r"(d)), i == 10);
  CHECK(6, asm("vmax.s32.u32.s32.sat.min %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(c), "r"(-20)), -20);
  CHECK(7, asm("vmax.s32.u32.s32.sat.max %0, %1, %2, %3;" : "=r"(i) : "r"(b), "r"(-33), "r"(9)), i == 9);

  return 0;
}

__device__ int vshl() {
  int i;
  unsigned u;
  CHECK(1, asm("vshl.s32.s32.u32.clamp %0, %1, %2;" : "=r"(i) : "r"(1), "r"(4)), i == 16);
  CHECK(2, asm("vshl.s32.u32.u32.clamp %0, %1, %2;" : "=r"(i) : "r"(1), "r"(33)), i == 0);
  CHECK(3, asm("vshl.u32.s32.u32.clamp %0, %1, %2;" : "=r"(u) : "r"(1), "r"(32)), u == 0);
  CHECK(4, asm("vshl.u32.u32.u32.clamp %0, %1, %2;" : "=r"(u) : "r"(1), "r"(2)), u == 4);
  CHECK(5, asm("vshl.s32.s32.u32.wrap %0, %1, %2;" : "=r"(i) : "r"(1), "r"(4)), i == 16);
  CHECK(6, asm("vshl.s32.u32.u32.wrap %0, %1, %2;" : "=r"(i) : "r"(1), "r"(33)), i == 2);
  CHECK(7, asm("vshl.u32.s32.u32.wrap %0, %1, %2;" : "=r"(u) : "r"(1), "r"(32)), u == 1);
  CHECK(8, asm("vshl.u32.u32.u32.wrap %0, %1, %2;" : "=r"(u) : "r"(1), "r"(2)), u == 4);
  return 0;
}

__device__ int vshr() {
  int i;
  unsigned u;
  CHECK(1, asm("vshr.s32.s32.u32.clamp %0, %1, %2;" : "=r"(i) : "r"(32), "r"(4)), i == 2);
  CHECK(2, asm("vshr.s32.u32.u32.clamp %0, %1, %2;" : "=r"(i) : "r"(1), "r"(33)), i == 0);
  CHECK(3, asm("vshr.u32.s32.u32.clamp %0, %1, %2;" : "=r"(u) : "r"(1), "r"(32)), u == 0);
  CHECK(4, asm("vshr.u32.u32.u32.clamp %0, %1, %2;" : "=r"(u) : "r"(3), "r"(2)), u == 0);
  CHECK(5, asm("vshr.s32.s32.u32.wrap %0, %1, %2;" : "=r"(i) : "r"(32), "r"(4)), i == 2);
  CHECK(6, asm("vshr.s32.u32.u32.wrap %0, %1, %2;" : "=r"(i) : "r"(1), "r"(33)), i == 0);
  CHECK(7, asm("vshr.u32.s32.u32.wrap %0, %1, %2;" : "=r"(u) : "r"(1), "r"(32)), u == 1);
  CHECK(8, asm("vshr.u32.u32.u32.wrap %0, %1, %2;" : "=r"(u) : "r"(32), "r"(2)), u == 8);
  return 0;
}

__device__ int vset() {
  int i;
  CHECK(1, asm("vset.s32.s32.eq %0, %1, %2;" : "=r"(i) : "r"(32), "r"(40)), i == 0);
  CHECK(2, asm("vset.s32.s32.ne %0, %1, %2;" : "=r"(i) : "r"(32), "r"(32)), i == 0);
  CHECK(3, asm("vset.s32.s32.lt %0, %1, %2;" : "=r"(i) : "r"(32), "r"(32)), i == 0);
  CHECK(4, asm("vset.s32.s32.le %0, %1, %2;" : "=r"(i) : "r"(30), "r"(32)), i == 1);
  CHECK(5, asm("vset.s32.s32.gt %0, %1, %2;" : "=r"(i) : "r"(32), "r"(31)), i == 1);
  CHECK(6, asm("vset.s32.s32.ge %0, %1, %2;" : "=r"(i) : "r"(32), "r"(32)), i == 1);
  CHECK(7, asm("vset.s32.s32.eq.add %0, %1, %2, %3;" : "=r"(i) : "r"(32), "r"(40), "r"(1)), i == 1);
  CHECK(8, asm("vset.s32.s32.ne.min %0, %1, %2, %3;" : "=r"(i) : "r"(32), "r"(32), "r"(3)), i == 0);
  CHECK(9, asm("vset.s32.s32.lt.max %0, %1, %2, %3;" : "=r"(i) : "r"(32), "r"(32), "r"(4)), i == 4);
  return 0;
}

// clang-format on

__global__ void test(int *ec) {
  {
    int res = vadd();
    if (res != 0) {
      *ec = res;
      return;
    }
  }
  {
    int res = vsub();
    if (res != 0) {
      *ec = res;
      return;
    }
  }
  {
    int res = vabsdiff();
    if (res != 0) {
      *ec = res;
      return;
    }
  }
  {
    int res = vmin();
    if (res != 0) {
      *ec = res;
      return;
    }
  }
  {
    int res = vmax();
    if (res != 0) {
      *ec = res;
      return;
    }
  }
  {
    int res = vshl();
    if (res != 0) {
      *ec = res;
      return;
    }
  }
  {
    int res = vshr();
    if (res != 0) {
      *ec = res;
      return;
    }
  }
  {
    int res = vset();
    if (res != 0) {
      *ec = res;
      return;
    }
  }
}

int main() {
  int *ec = nullptr;
  hipMallocManaged(&ec, sizeof(int));
  *ec = 0;
  test<<<1, 1>>>(ec);
  hipDeviceSynchronize();
  if (*ec != 0) {
    printf("Test failed %d\n", *ec);
  } else {
    printf("Test pass\n");
  }
  return 0;
}
