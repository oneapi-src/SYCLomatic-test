// ====------ module-main.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <string>
int main(){
    hipModule_t M;
    hipFunction_t F;
    std::string Path, FunctionName, Data;
    FunctionName = "foo";
    hipModuleLoad(&M, Path.c_str());
    hipModuleLoadData(&M, Data.c_str());
    hipModuleGetFunction(&F, M, FunctionName.c_str());
    float *param[2] = {0};
    hipMalloc(&param[0], sizeof(float));
    hipMalloc(&param[1], sizeof(float));
    hipModuleLaunchKernel(F, 1, 1, 1, 1, 1, 1, 10, 0, (void**)param, nullptr);
    hipTexRef tex;
    hipModuleGetTexRef(&tex, M, "tex");
    hipModuleUnload(M);
    hipDeviceSynchronize();
    hipFree(param[0]);
    hipFree(param[1]);
    return 0;
}
