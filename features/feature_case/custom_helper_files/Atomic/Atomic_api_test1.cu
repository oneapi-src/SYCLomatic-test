
#include <hip/hip_runtime.h>
// ====------ Atomic_api_test1.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Atomic/api_test1_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Atomic/api_test1_out/MainSourceFiles.yaml | wc -l > %T/Atomic/api_test1_out/count.txt
// RUN: FileCheck --input-file %T/Atomic/api_test1_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Atomic/api_test1_out

// CHECK: 2
// TEST_FEATURE: Atomic_atomic_compare_exchange_strong

__global__ void test(int *data) {
  int inc = 1;


  atomicCAS(&data[7], inc - 1, inc);
}
int main() {
  return 0;
}
