
#include <hip/hip_runtime.h>
// ====------ Memory_api_test39.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test39_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test39_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test39_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test39_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test39_out

// CHECK: 29
// TEST_FEATURE: Memory_async_dpct_memcpy
// TEST_FEATURE: Memory_async_dpct_memcpy_2d
// TEST_FEATURE: Memory_async_dpct_memcpy_3d

int main() {
  hipMemcpy3DParms parms;
  hipMemcpy3DAsync(&parms);
  return 0;
}
