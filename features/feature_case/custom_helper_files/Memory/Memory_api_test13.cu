// ====------ Memory_api_test13.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test13_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test13_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test13_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test13_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test13_out

// CHECK: 10
// TEST_FEATURE: Memory_mem_mgr_is_device_ptr


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  hipblasHandle_t handle;
  float *x_S;
  int *result;
  hipblasIsamax(handle, 10, x_S, 1, result);
  return 0;
}
