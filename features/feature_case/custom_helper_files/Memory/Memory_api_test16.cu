
#include <hip/hip_runtime.h>
// ====------ Memory_api_test16.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test16_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test16_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test16_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test16_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test16_out

// CHECK: 45
// TEST_FEATURE: Memory_constant_memory_alias

__constant__ float A[1024];

int main() {
  return 0;
}
