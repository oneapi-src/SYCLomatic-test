
#include <hip/hip_runtime.h>
// ====------ Memory_api_test5.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test5_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test5_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test5_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test5_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test5_out

// CHECK: 16
// TEST_FEATURE: Memory_dpct_malloc

int main() {
  float* a;
  hipMalloc(&a, 4);
  return 0;
}
