
#include <hip/hip_runtime.h>
// ====------ Memory_api_test30.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test30_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test30_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test30_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test30_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test30_out

// CHECK: 40
// TEST_FEATURE: Memory_device_memory_get_ptr_q

__constant__ float constData[4];

int main() {
  hipStream_t stream;
  float* host;
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), host, 1, 3, hipMemcpyHostToDevice, stream);
  return 0;
}
