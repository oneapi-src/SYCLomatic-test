
#include <hip/hip_runtime.h>
// ====------ Image_api_test35.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Image/api_test35_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test35_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test35_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test35_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test35_out

// CHECK: 44
// usm version of 21
// TEST_FEATURE: Image_image_matrix_to_pitched_data

int main() {
  hipArray_t a1;
  hipArray* a2;
  size_t width, height, woffset, hoffset;
  hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height, hipMemcpyDeviceToHost);
  return 0;
}