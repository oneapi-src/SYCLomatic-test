// ====------ Util_api_test11.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Util/api_test11_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test11_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test11_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test11_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test11_out

// CHECK: 24

// TEST_FEATURE: Util_matrix_mem_copy_T


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  hipblasHandle_t handle;
  float* a;
  float alpha;
  hipblasStrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, 4, 4, &alpha, a, 4, a, 4, a, 4);
  return 0;
}
