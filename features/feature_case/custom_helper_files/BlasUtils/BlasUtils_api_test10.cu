// ====------ BlasUtils_api_test10.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test10_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test10_out

// CHECK: 34


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_geqrf_batch_wrapper

int main() {
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int lda = 275;

  float **Aarray_S = 0;
  float **TauArray_S = 0;
  int *infoArray = 0;
  int batchSize = 10;

  hipblasSgeqrfBatched(handle, m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize);
  return 0;
}
