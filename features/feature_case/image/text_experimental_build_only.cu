// ===-------- text_experimental_build_only.cu ------- *- CUDA -* ---------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//

#include <hip/hip_runtime.h>

__global__ void
CppLanguageExtensions_TextureFunctions(hipTextureObject_t tex) {
  int i = 1, t = 1;
  float j = 1, k = 1, l = 1, m = 1;
  tex1Dfetch<float4>(tex, i);
  tex1D<short2>(tex, i);
  tex1DLod<ushort2>(tex, i, t);
  tex2D<int1>(tex, j, k);
  tex2DLod<uint1>(tex, j, k, l);
  tex3D<char4>(tex, j, k, l);
  tex3DLod<uchar4>(tex, j, k, l, m);
  // tex1DLayered<uchar2>(tex, i, t); // TODO: need support.
  // tex2DLayered<uint2>(tex, j, k, t); // TODO: need support.
}

void Runtime_MemoryManagement() {
  hipChannelFormatDesc d;
  hipExtent e;
  unsigned int u;
  hipArray_t a = nullptr;
  hipMipmappedArray_t m;
  hipPitchedPtr p;
  size_t s = 1;
  void *v;
  hipMemcpyKind k = hipMemcpyDefault;
  hipMemcpy3DParms pm;
  int i = 1;
  hipArrayGetInfo(&d, &e, &u, a);
  hipFreeArray(a);
  hipFreeMipmappedArray(m);
  hipGetMipmappedArrayLevel(&a, m, u);
  hipMalloc3D(&p, e);
  hipMalloc3DArray(&a, &d, e, u);
  hipMallocArray(&a, &d, s, s, u);
  hipMallocMipmappedArray(&m, &d, e, u, i);
  hipMallocPitch(&v, &s, s, s);
  hipMemcpy2D(v, s, v, s, s, s, k);
  hipMemcpy2DArrayToArray(a, s, s, a, s, s, s, s, k);
  hipMemcpy2DAsync(v, s, v, s, s, s, k);
  hipMemcpy2DFromArray(v, s, a, s, s, s, s, k);
  hipMemcpy2DFromArrayAsync(v, s, a, s, s, s, s, k);
  hipMemcpy2DToArray(a, s, s, v, s, s, s, k);
  hipMemcpy2DToArrayAsync(a, s, s, v, s, s, s, k);
  hipMemcpy3D(&pm);
  hipMemcpy3DAsync(&pm);
  hipMemset2D(v, s, i, s, s);
  hipMemset2DAsync(v, s, i, s, s);
  hipMemset3D(p, i, e);
  hipMemset3DAsync(p, i, e);
}

void Runtime_TextureObjectManagement() {
  int i = 1;
  hipChannelFormatKind k = hipChannelFormatKindSigned;
  hipTextureObject_t o;
  hipResourceDesc r;
  hipTextureDesc t;
  // cudaResourceViewDesc v; // TODO: need support.
  hipArray_t a = nullptr;
  hipChannelFormatDesc d;
  hipCreateChannelDesc(i, i, i, i, k);
  hipCreateTextureObject(&o, &r, &t, nullptr /*&v*/);
  hipDestroyTextureObject(o);
  hipGetChannelDesc(&d, a);
  hipGetTextureObjectResourceDesc(&r, o);
  hipGetTextureObjectTextureDesc(&t, o);
}

void Driver_MemoryManagement() {
  hipArray_t a;
  HIP_ARRAY_DESCRIPTOR D;
  hip_Memcpy2D C2;
  hipStream_t cs;
  HIP_MEMCPY3D C3;
  size_t s;
  hipDeviceptr_t d;
  void *v;
  hipArrayCreate(&a, &D);
  hipArrayDestroy(a);
  hipMemcpyParam2D(&C2);
  hipMemcpyParam2DAsync(&C2, cs);
  hipDrvMemcpy3D(&C3);
  hipDrvMemcpy3DAsync(&C3, cs);
  hipMemcpyAtoA(a, s, a, s, s);
  hipMemcpyAtoD(d, a, s, s);
  hipMemcpyAtoH(&v, a, s, s);
  hipMemcpyAtoHAsync(&v, a, s, s, cs);
  hipMemcpyDtoA(a, s, d, s);
  hipMemcpyDtoD(d, d, s);
  hipMemcpyDtoDAsync(d, d, s, cs);
  hipMemcpyDtoH(v, d, s);
  hipMemcpyDtoHAsync(v, d, s, cs);
  hipMemcpyHtoA(a, s, v, s);
  hipMemcpyHtoAAsync(a, s, v, s, cs);
  hipMemcpyHtoD(d, v, s);
  hipMemcpyHtoDAsync(d, v, s, cs);
}

void Driver_TextureObjectManagement() {
  hipTextureObject_t o;
  HIP_RESOURCE_DESC R;
  HIP_TEXTURE_DESC T;
  // CUDA_RESOURCE_VIEW_DESC V; // TODO: need support.
  // cuTexObjectCreate(&o, &R, &T, nullptr /*&V*/); // TODO: need support.
  hipTexObjectDestroy(o);
  // cuTexObjectGetResourceDesc(&R, o); // TODO: need support.
  // cuTexObjectGetTextureDesc(&T, o);  // TODO: need support.
}

int main() {
  Runtime_MemoryManagement();
  Runtime_TextureObjectManagement();
  Driver_MemoryManagement();
  Driver_TextureObjectManagement();
  hipTextureObject_t tex;
  CppLanguageExtensions_TextureFunctions<<<1, 1>>>(tex);
  return 0;
}
